#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "util.h"
#define N 10000000
#define MAX_ERR 1e-6


__global__ void vector_add(float *out, float *a, float *b, int n) {
    int tid = global_thread_id() ;
    int stride = block_size();
    for(int i = tid; i < n; i += stride){
       out[i] = a[i] + b[i];
    }
}
int main(){
    float *a, *b, *out;
    float *d_a, *d_b, *d_out; 

    // Allocate host memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize host arrays
    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Allocate device memory
    gpuErrchk(hipMalloc((void**)&d_a, sizeof(float) * N));
    gpuErrchk(hipMalloc((void**)&d_b, sizeof(float) * N));
    gpuErrchk(hipMalloc((void**)&d_out, sizeof(float) * N));

    // Transfer data from host to device memory
    gpuErrchk(hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice));

    // Executing kernel 
    int block_size = 256;
    int grid_size = 1;
    double start = get_time();

    vector_add<<<grid_size,block_size>>>(d_out, d_a, d_b, N);
    // Transfer data back to host memory
    hipDeviceSynchronize();

    double end = get_time();


    gpuErrchk(hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost));

    // Verification
    for(int i = 0; i < N; i++){
    
        // printf("out = %f\n", out[i]);
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }
    // printf("out[0] = %f\n", out[0]);

    auto cost_time = end - start;

    printf("PASSED, computation time GPU_one_grid = %f secs\n", cost_time);

    // Deallocate device memory
    gpuErrchk(hipFree(d_a));
    gpuErrchk(hipFree(d_b));
    gpuErrchk(hipFree(d_out));

    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);
}
