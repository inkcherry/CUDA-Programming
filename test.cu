#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 100
#define MAX_ERR 1e-6
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i ++){
        out[i] = a[i] + b[i];
    }
}
int main(){
    float *a, *b, *out;
    float *d_a, *d_b, *d_out; 

    // Allocate host memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize host arrays
    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Allocate device memory
    gpuErrchk(hipMalloc((void**)&d_a, sizeof(float) * N));
    gpuErrchk(hipMalloc((void**)&d_b, sizeof(float) * N));
    gpuErrchk(hipMalloc((void**)&d_out, sizeof(float) * N));

    // Transfer data from host to device memory
    gpuErrchk(hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice));

    // Executing kernel 
    vector_add<<<1,1>>>(d_out, d_a, d_b, N);
    
    // Transfer data back to host memory
    gpuErrchk(hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost));

    // Verification
    for(int i = 0; i < N; i++){
    
        printf("out = %f\n", out[i]);
        // assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }
    // printf("out[0] = %f\n", out[0]);
    printf("PASSED\n");

    // Deallocate device memory
    gpuErrchk(hipFree(d_a));
    gpuErrchk(hipFree(d_b));
    gpuErrchk(hipFree(d_out));

    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);
}
